#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        printf("No CUDA-capable devices found.\n");
        return 1;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // Query device 0 (assuming single GPU)

    printf("Device Name: %s\n", deviceProp.name);
    printf("Max Threads Per Block: %d\n", deviceProp.maxThreadsPerBlock);

    return 0;
}